#include "hip/hip_runtime.h"
// sorting.cu
#include "sorting.cuh"
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/gather.h>
#include <thrust/copy.h>
#include <hip/hip_runtime.h>

Sorting::Sorting(ParticleContainer& pc_, FieldContainer& fc_)
    : pc(&pc_), fc(&fc_), n_particles(pc_.n_particles),
      nx(fc_.nx), ny(fc_.ny),
      xmin(fc_.xmin), ymin(fc_.ymin),
      dx(fc_.dx), dy(fc_.dy)
{
    d_cell_indices.resize(n_particles);
    d_particle_indices.resize(n_particles);
    temp_buffer.resize(n_particles);
}

void Sorting::sort_particles_by_cell() {
    int threads = 256;
    int blocks = (n_particles + threads - 1) / threads;

    compute_cell_indices<<<blocks, threads>>>(
        pc->d_x, pc->d_y,
        thrust::raw_pointer_cast(d_cell_indices.data()),
        n_particles,
        nx, ny,
        xmin, ymin,
        dx, dy
    );
    hipDeviceSynchronize();

    thrust::sequence(d_particle_indices.begin(), d_particle_indices.end());

    thrust::sort_by_key(
        d_cell_indices.begin(), d_cell_indices.end(),
        d_particle_indices.begin()
    );

    reorder_particle_data();
}

void Sorting::reorder_particle_data() {

    auto gather_and_copy = [&](float* src_ptr) {
        thrust::device_ptr<float> src(src_ptr);
        thrust::gather(d_particle_indices.begin(), d_particle_indices.end(), src, temp_buffer.begin());
        thrust::copy(temp_buffer.begin(), temp_buffer.end(), src);
    };

    gather_and_copy(pc->d_x);
    gather_and_copy(pc->d_y);
    gather_and_copy(pc->d_vx);
    gather_and_copy(pc->d_vy);
    gather_and_copy(pc->d_w);
}

__global__ void compute_cell_indices(
    float* d_x, float* d_y,
    int* d_cell_indices,
    int n,
    int nx, int ny,
    float xmin, float ymin,
    float dx, float dy
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    int ix = (d_x[i] - xmin) / dx;
    int iy = (d_y[i] - ymin) / dy;

    ix = max(0, min(ix, nx - 1));
    iy = max(0, min(iy, ny - 1));

    d_cell_indices[i] = iy * nx + ix;
}
