#include "hip/hip_runtime.h"
#pragma once
#include "SortedDepositor.h"

__global__ void deposit_density_2d_sorted(...) { ... }
__global__ void deposit_velocity_2d_sorted(...) { ... }
__global__ void deposit_temperature_2d_sorted(...) { ... }
__global__ void deposit_density_2d_VR_sorted(...) { ... }
__global__ void deposit_velocity_2d_VR_sorted(...) { ... }
__global__ void deposit_temperature_2d_VR_sorted(...) { ... }

void BruteDepositor::deposit(ParticleContainer& pc, FieldContainer& fc, Sorting& /*sorter*/) {

    launch(deposit_density_2d_sorted, blocks, threads, pc.d_x, pc.d_y, fc.d_N,
           n_particles, N_GRID_X, N_GRID_Y, Lx, Ly);
    hipDeviceSynchronize();

    launch(deposit_velocity_2d_sorted, blocks, threads, pc.d_x, pc.d_y, fc.d_N,
           pc.d_vx, pc.d_vy, fc.d_Ux, fc.d_Uy, n_particles, N_GRID_X, N_GRID_Y, Lx, Ly);
    hipDeviceSynchronize();

    launch(deposit_temperature_2d_sorted, blocks, threads, pc.d_x, pc.d_y, fc.d_N,
           pc.d_vx, pc.d_vy, fc.d_Ux, fc.d_Uy, fc.d_T, n_particles, N_GRID_X, N_GRID_Y, Lx, Ly);
    hipDeviceSynchronize();

    launch(deposit_density_2d_VR_sorted, blocks, threads, pc.d_x, pc.d_y, pc.d_w, fc.d_N, fc.d_NVR,
           n_particles, N_GRID_X, N_GRID_Y, Lx, Ly);
    hipDeviceSynchronize();

    launch(deposit_velocity_2d_VR_sorted, blocks, threads, pc.d_x, pc.d_y, pc.d_vx, pc.d_vy,
           pc.d_w, fc.d_UxVR, fc.d_UyVR, fc.d_NVR, n_particles, N_GRID_X, N_GRID_Y, Lx, Ly);
    hipDeviceSynchronize();

    launch(deposit_temperature_2d_VR_tiled, blocks, threads, pc.d_x, pc.d_y, pc.d_vx, pc.d_vy,
           pc.d_w, fc.d_N, fc.d_NVR, fc.d_UxVR, fc.d_UyVR, fc.d_TVR,
           n_particles, N_GRID_X, N_GRID_Y, Lx, Ly);
    hipDeviceSynchronize();
}

__global__ void copy_counts_to_density(
    const int* __restrict__ cell_counts,
    float* __restrict__ density,
    int num_cells
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_cells) return;
    density[i] = static_cast<float>(cell_counts[i]);
}

void deposit_density_2d_sorted(Sorting& sorter, hipStream_t stream = 0) {
    int num_cells = sorter.nx * sorter.ny;

    // cell_counts was already computed in sort_particles_and_compute_density()
    // so we just copy it into the density field.
    copy_counts_to_density<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        sorter.d_cell_counts,
        sorter.fc->d_N,
        num_cells
    );

    // optional: sync if you need immediate access to density
    hipStreamSynchronize(stream);
}

__global__ void deposit_velocity_2d_sorted(
    const float* __restrict__ vx,
    const float* __restrict__ vy,
    const int* __restrict__ d_cell_offsets,
    float* __restrict__ Ux,
    float* __restrict__ Uy,
    int num_cells
) {
    int cell = blockIdx.x * blockDim.x + threadIdx.x;
    if (cell >= num_cells) return;

    // Get start and end index for this cell
    int start = d_cell_offsets[cell];
    int end   = d_cell_offsets[cell + 1];

    float sum_vx = 0.0f;
    float sum_vy = 0.0f;
    int count = end - start;

    // Sum over particles in this cell
    for (int i = start; i < end; i++) {
        sum_vx += vx[i];
        sum_vy += vy[i];
    }

    // Store average velocity (avoid division by zero)
    if (count > 0) {
        Ux[cell] = sum_vx / count;
        Uy[cell] = sum_vy / count;
    } else {
        Ux[cell] = 0.0f;
        Uy[cell] = 0.0f;
    }
}

// T = ( <(vx-Ux)^2 + (vy-Uy)^2> ) / (2 * kb/m)
__global__ void deposit_temperature_2d_sorted(
    const float* __restrict__ vx,
    const float* __restrict__ vy,
    const int*   __restrict__ d_cell_offsets, // size: num_cells + 1
    const float* __restrict__ Ux,
    const float* __restrict__ Uy,
    float* T,
    int num_cells
) {
    int cell = blockIdx.x * blockDim.x + threadIdx.x;
    if (cell >= num_cells) return;

    int start = d_cell_offsets[cell];
    int end   = d_cell_offsets[cell + 1]; // exclusive

    float ux = Ux[cell];
    float uy = Uy[cell];
    float temp_sum = 0.0f;
    int npart = end - start;

    for (int i = start; i < end; ++i) {
        float dvx = vx[i] - ux;
        float dvy = vy[i] - uy;
        temp_sum += dvx * dvx + dvy * dvy;
    }

    T[cell] = (npart > 0) ? temp_sum / (2.0f * kb/m * npart) : 0.0f;
}



__global__ void deposit_density_2d_VR_sorted(
    const float* __restrict__ w,          // particle weights
    const int*   __restrict__ d_cell_offsets, // per-cell start indices (size num_cells+1)
    float* NVR,                           // output: variance-reduced density
    int num_cells,
    int n_particles
) {
    int cell = blockIdx.x * blockDim.x + threadIdx.x;
    if (cell >= num_cells) return;

    int start = d_cell_offsets[cell];
    int end   = d_cell_offsets[cell + 1]; // exclusive
    int npart = end - start;

    float Navg = float(n_particles) / float(num_cells);
    float sum = 0.0f;

    for (int i = start; i < end; ++i) {
        sum += 1.0f - w[i];
    }

    NVR[cell] = (npart > 0) ? Navg + sum : Navg;
}




__global__ void deposit_velocity_2d_VR_sorted(
    const float* __restrict__ vx,
    const float* __restrict__ vy,
    const float* __restrict__ w,
    const int*   __restrict__ d_cell_offsets, // start indices of particles per cell
    const float* __restrict__ NVR,            // number of particles per cell / density
    float* UxVR,                              // output: x-velocity per cell
    float* UyVR,                              // output: y-velocity per cell
    int num_cells
) {
    int cell = blockIdx.x * blockDim.x + threadIdx.x;
    if (cell >= num_cells) return;

    int start = d_cell_offsets[cell];
    int end   = d_cell_offsets[cell + 1]; // exclusive
    int npart = end - start;

    float sum_vx = 0.0f;
    float sum_vy = 0.0f;

    for (int i = start; i < end; ++i) {
        float factor = 1.0f - w[i];
        sum_vx += vx[i] * factor;
        sum_vy += vy[i] * factor;
    }

    // Avoid division by zero
    if (npart > 0) {
        UxVR[cell] = sum_vx / NVR[cell];
        UyVR[cell] = sum_vy / NVR[cell];
    } else {
        UxVR[cell] = 0.0f;
        UyVR[cell] = 0.0f;
    }
}





__global__ void deposit_temperature_2d_VR_sorted(
    const float* __restrict__ vx,
    const float* __restrict__ vy,
    const float* __restrict__ w,
    const float* __restrict__ UxVR,
    const float* __restrict__ UyVR,
    const int*   __restrict__ d_cell_offsets, // start indices of particles per cell
    const float* __restrict__ NVR,            // VR density
    float* TVR,                               // output: VR temperature per cell
    int num_cells
) {
    int cell = blockIdx.x * blockDim.x + threadIdx.x;
    if (cell >= num_cells) return;

    int start = d_cell_offsets[cell];
    int end   = d_cell_offsets[cell + 1]; // exclusive
    int npart = end - start;

    float temp_sum = 0.0f;

    // Navg for variance reduction
    float Navg = 0.0f;
    if (num_cells > 0) {
        Navg = float(d_cell_offsets[num_cells]) / float(num_cells); // total_particles / num_cells
    }

    float energy;

    for (int i = start; i < end; ++i) {
        float dvx = vx[i] - UxVR[cell];
        float dvy = vy[i] - UyVR[cell];
        energy = (dvx*dvx + dvy*dvy) * 0.5f * (1.0f - w[i]);
    }

    temp_sum = energy / npart / NVR[cell] / (kb/m); // divide by VR density

    // Add eq. term
    if (npart > 0.0f) {
        temp_sum += Navg/(kb/m)/NVR[cell];
    }

    TVR[cell] = temp_sum;
}