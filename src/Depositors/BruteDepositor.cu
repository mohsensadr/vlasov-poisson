#include "hip/hip_runtime.h"
#include "BruteDepositor.h"

__global__ void deposit_density_2d(float *x, float *y, float *N, int n_particles,
            int N_GRID_X, int N_GRID_Y,
            float Lx, float Ly
  );

__global__ void deposit_velocity_2d(float *x, float *y, float *N, float *vx, float *vy, float *Ux, float *Uy, int n_particles,
            int N_GRID_X, int N_GRID_Y,
            float Lx, float Ly
  );

__global__ void deposit_temperature_2d(float *x, float *y, float *N, float *vx, float *vy, float *Ux, float *Uy, float *T, int n_particles,
            int N_GRID_X, int N_GRID_Y,
            float Lx, float Ly
  );

__global__ void deposit_density_2d_VR(float *x, float *y, float *w, float *N, float *NVR, int n_particles,
            int N_GRID_X, int N_GRID_Y,
            float Lx, float Ly
  );

__global__ void deposit_velocity_2d_VR(float *x, float *y, float *vx, float*vy, float *w,
            float *UxVR, float *UyVR, float *NVR, int n_particles,
            int N_GRID_X, int N_GRID_Y,
            float Lx, float Ly
  );

__global__ void deposit_temperature_2d_VR(float *x, float *y, float *vx, float *vy, float *w, float *N, float *NVR, float *UxVR, float *UyVR, float *TVR, int n_particles,
            int N_GRID_X, int N_GRID_Y,
            float Lx, float Ly
  );

void BruteDepositor::deposit(ParticleContainer& pc, FieldContainer& fc, Sorting& /*sorter*/) {
    int n_particles = N_PARTICLES;
    dim3 threads(256);
    dim3 blocks((n_particles + 255) / 256);

    launch(deposit_density_2d, blocks, threads, pc.d_x, pc.d_y, fc.d_N,
           n_particles, N_GRID_X, N_GRID_Y, Lx, Ly);

    launch(deposit_velocity_2d, blocks, threads, pc.d_x, pc.d_y, fc.d_N,
           pc.d_vx, pc.d_vy, fc.d_Ux, fc.d_Uy, n_particles, N_GRID_X, N_GRID_Y, Lx, Ly);

    launch(deposit_temperature_2d, blocks, threads, pc.d_x, pc.d_y, fc.d_N,
           pc.d_vx, pc.d_vy, fc.d_Ux, fc.d_Uy, fc.d_T, n_particles, N_GRID_X, N_GRID_Y, Lx, Ly);

    launch(deposit_density_2d_VR, blocks, threads, pc.d_x, pc.d_y, pc.d_w, fc.d_N, fc.d_NVR,
           n_particles, N_GRID_X, N_GRID_Y, Lx, Ly);

    launch(deposit_velocity_2d_VR, blocks, threads, pc.d_x, pc.d_y, pc.d_vx, pc.d_vy,
           pc.d_w, fc.d_UxVR, fc.d_UyVR, fc.d_NVR, n_particles, N_GRID_X, N_GRID_Y, Lx, Ly);

    launch(deposit_temperature_2d_VR, blocks, threads, pc.d_x, pc.d_y, pc.d_vx, pc.d_vy,
           pc.d_w, fc.d_N, fc.d_NVR, fc.d_UxVR, fc.d_UyVR, fc.d_TVR,
           n_particles, N_GRID_X, N_GRID_Y, Lx, Ly);
}

__global__ void deposit_density_2d(float *x, float *y, float *N, int n_particles,
            int N_GRID_X, int N_GRID_Y,
            float Lx, float Ly
    ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_particles) {
        int ix = int(x[i] / Lx * N_GRID_X) % N_GRID_X;
        int iy = int(y[i] / Ly * N_GRID_Y) % N_GRID_Y;
        int idx = ix + iy * N_GRID_X;
        atomicAdd(&N[idx], 1.0f);
    }
}

__global__ void deposit_velocity_2d(float *x, float *y, float *N, float *vx, float *vy, float *Ux, float *Uy, int n_particles,
            int N_GRID_X, int N_GRID_Y,
            float Lx, float Ly
    ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_particles) {
        int ix = int(x[i] / Lx * N_GRID_X) % N_GRID_X;
        int iy = int(y[i] / Ly * N_GRID_Y) % N_GRID_Y;
        int idx = ix + iy * N_GRID_X;
        atomicAdd(&Ux[idx], vx[i]/N[idx]);
        atomicAdd(&Uy[idx], vy[i]/N[idx]);
    }
}

__global__ void deposit_temperature_2d(float *x, float *y, float *N, float *vx, float *vy, float *Ux, float *Uy, float *T, int n_particles,
            int N_GRID_X, int N_GRID_Y,
            float Lx, float Ly
    ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_particles) {
        int ix = int(x[i] / Lx * N_GRID_X) % N_GRID_X;
        int iy = int(y[i] / Ly * N_GRID_Y) % N_GRID_Y;
        int idx = ix + iy * N_GRID_X;
        float energy = (vx[i]-Ux[idx])*(vx[i]-Ux[idx]);
        energy += (vy[i]-Uy[idx])*(vy[i]-Uy[idx]);
        atomicAdd(&T[idx], energy/N[idx]/(2.0f*kb/m));
    }
}

__global__ void deposit_density_2d_VR(float *x, float *y, float *w, float *N, float *NVR, int n_particles,
            int N_GRID_X, int N_GRID_Y,
            float Lx, float Ly
    ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_particles) {
        float Navg = (1.0f*n_particles) / (1.0f*N_GRID_X*N_GRID_Y);
        int ix = int(x[i] / Lx * N_GRID_X) % N_GRID_X;
        int iy = int(y[i] / Ly * N_GRID_Y) % N_GRID_Y;
        int idx = ix + iy * N_GRID_X;
        atomicAdd(&NVR[idx], Navg/N[idx] + 1.0f - w[i] );
    }
}

__global__ void deposit_velocity_2d_VR(float *x, float *y, float *vx, float*vy, float *w,
            float *UxVR, float *UyVR, float *NVR, int n_particles,
            int N_GRID_X, int N_GRID_Y,
            float Lx, float Ly
    ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_particles) {
        int ix = int(x[i] / Lx * N_GRID_X) % N_GRID_X;
        int iy = int(y[i] / Ly * N_GRID_Y) % N_GRID_Y;
        int idx = ix + iy * N_GRID_X;
        atomicAdd(&UxVR[idx], vx[i] * ( 1.0f - w[i] ) / NVR[idx] );
        atomicAdd(&UyVR[idx], vy[i] * ( 1.0f - w[i] ) / NVR[idx] );
    }
}

__global__ void deposit_temperature_2d_VR(float *x, float *y, float *vx, float *vy, float *w, float *N, float *NVR, float *UxVR, float *UyVR, float *TVR, int n_particles,
            int N_GRID_X, int N_GRID_Y,
            float Lx, float Ly
    ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_particles) {
        float Navg = (1.0f*n_particles) / (1.0f*N_GRID_X*N_GRID_Y);
        int ix = int(x[i] / Lx * N_GRID_X) % N_GRID_X;
        int iy = int(y[i] / Ly * N_GRID_Y) % N_GRID_Y;
        int idx = ix + iy * N_GRID_X;
        float energy = (vx[i]-UxVR[idx])*(vx[i]-UxVR[idx]);
        energy += (vy[i]-UyVR[idx])*(vy[i]-UyVR[idx]);
        float ans = Navg/(kb/m)/NVR[idx]/N[idx] + ( energy*(1.0f-w[i])/2.0f ) / (kb/m) / NVR[idx];
        atomicAdd(&TVR[idx], ans);
    }
}