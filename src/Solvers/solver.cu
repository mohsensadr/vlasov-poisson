#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <hipfft/hipfft.h>

#include "Solvers/solver.cuh"
#include "Constants/constants.hpp"

// ---------------------------------------------
// Utility: check CUDA errors
// ---------------------------------------------
#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        printf("CUDA error: %s\n", hipGetErrorString(err)); \
        exit(-1); \
    }

// ---------------------------------------------
// Kernel: scale spectrum (solve in Fourier space)
// ---------------------------------------------
__global__ void scale_kernel(const hipfftComplex* rhs_hat, hipfftComplex* phi_hat,
                             int NX, int NY, float_type dx, float_type dy) {
    int kx = blockIdx.x * blockDim.x + threadIdx.x;
    int ky = blockIdx.y * blockDim.y + threadIdx.y;
    if (kx >= NX/2+1 || ky >= NY) return;

    int idx = ky * (NX/2+1) + kx;

    float_type kx_val = 2.0 * M_PI * ((kx <= NX/2) ? kx : kx - NX) / (NX * dx);
    float_type ky_val = 2.0 * M_PI * ((ky <= NY/2) ? ky : ky - NY) / (NY * dy);

    float_type denom = (kx_val*kx_val + ky_val*ky_val);

    if (denom > 1e-14) {
        phi_hat[idx].x = rhs_hat[idx].x / denom;
        phi_hat[idx].y = rhs_hat[idx].y / denom;
    } else {
        // Zero-frequency mode -> set to 0 (fix nullspace)
        phi_hat[idx].x = 0.0;
        phi_hat[idx].y = 0.0;
    }
}

// ---------------------------------------------
// Kernel: scale real field (normalize after iFFT)
// ---------------------------------------------
__global__ void scale_real_kernel(float_type* arr, float_type alpha, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) arr[idx] *= alpha;
}

// ---------------------------------------------
// Kernel: compute residual r = Laplacian(phi) + N
// ---------------------------------------------
__global__ void compute_residual_kernel(const float_type *phi, const float_type *d_rhs, float_type *residual,
                                        int NX, int NY, float_type dx, float_type dy) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= NX || j >= NY) return;

    int ip = (i + 1) % NX;
    int im = (i - 1 + NX) % NX;
    int jp = (j + 1) % NY;
    int jm = (j - 1 + NY) % NY;

    int idx    = j * NX + i;
    int idx_im = j * NX + im;
    int idx_ip = j * NX + ip;
    int idx_jm = jm * NX + i;
    int idx_jp = jp * NX + i;

    float_type lap = (phi[idx_im] - 2.0*phi[idx] + phi[idx_ip]) / (dx*dx)
              + (phi[idx_jm] - 2.0*phi[idx] + phi[idx_jp]) / (dy*dy);

    residual[idx] = lap + d_rhs[idx];
}

// ---------------------------------------------
// Kernel: reduction for L2 norm
// ---------------------------------------------
__global__ void l2_reduce_kernel(const float_type* vec, float_type* block_sums, int N) {
    extern __shared__ float_type sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    float_type sum = 0.0;
    if (idx < N) {
        float_type v = vec[idx];
        sum += v*v;
    }
    if (idx + blockDim.x < N) {
        float_type v = vec[idx + blockDim.x];
        sum += v*v;
    }
    sdata[tid] = sum;
    __syncthreads();

    for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s) sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    if (tid == 0) block_sums[blockIdx.x] = sdata[0];
}

float_type compute_l2_norm(const float_type* d_vec, int N, int threads=256) {
    int blocks = (N + threads*2 - 1) / (threads*2);
    float_type* d_block_sums;
    CUDA_CHECK(hipMalloc(&d_block_sums, blocks * sizeof(float_type)));

    l2_reduce_kernel<<<blocks, threads, threads*sizeof(float_type)>>>(d_vec, d_block_sums, N);
    CUDA_CHECK(hipDeviceSynchronize());

    float_type* h_block_sums = new float_type[blocks];
    CUDA_CHECK(hipMemcpy(h_block_sums, d_block_sums, blocks*sizeof(float_type), hipMemcpyDeviceToHost));

    float_type sum = 0.0;
    for (int i=0; i<blocks; i++) sum += h_block_sums[i];

    delete[] h_block_sums;
    hipFree(d_block_sums);

    return sqrt(sum);
}

// ---------------------------------------------
// Direct Poisson solver using cuFFT
// ---------------------------------------------
void poisson_fft_solver(int NX, int NY, float_type dx, float_type dy,
                        float_type* d_rhs, float_type* d_phi) {
    hipfftHandle planR2C, planC2R;
    hipfftPlan2d(&planR2C, NY, NX, HIPFFT_R2C);
    hipfftPlan2d(&planC2R, NY, NX, HIPFFT_C2R);

    int nComplex = NY * (NX/2 + 1);
    hipfftComplex* d_rhs_hat;
    hipfftComplex* d_phi_hat;
    CUDA_CHECK(hipMalloc(&d_rhs_hat, sizeof(hipfftComplex)*nComplex));
    CUDA_CHECK(hipMalloc(&d_phi_hat, sizeof(hipfftComplex)*nComplex));

    // Forward FFT
    hipfftExecR2C(planR2C, (hipfftReal*)d_rhs, d_rhs_hat);

    // Solve in spectral space
    dim3 threads2D(16,16);
    dim3 blocks2D((NX/2+1 + 15)/16, (NY+15)/16);
    scale_kernel<<<blocks2D, threads2D>>>(d_rhs_hat, d_phi_hat, NX, NY, dx, dy);

    // Inverse FFT
    hipfftExecC2R(planC2R, d_phi_hat, (hipfftReal*)d_phi);

    // Normalize
    int size = NX*NY;
    float_type alpha = 1.0 / float_type(size);
    scale_real_kernel<<<(size+255)/256, 256>>>(d_phi, alpha, size);

    hipfftDestroy(planR2C);
    hipfftDestroy(planC2R);
    hipFree(d_rhs_hat);
    hipFree(d_phi_hat);
}

static __device__ int periodic_index(int i, int N) {
    return (i + N) % N;
}

__global__ void compute_electric_field_kernel_periodic(const float_type *phi, float_type *Ex, float_type *Ey,
                                              int N_GRID_X, int N_GRID_Y, float_type dx, float_type dy) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N_GRID_X && j < N_GRID_Y) {
        int ip = periodic_index(i + 1, N_GRID_X);
        int im = periodic_index(i - 1, N_GRID_X);
        int jp = periodic_index(j + 1, N_GRID_Y);
        int jm = periodic_index(j - 1, N_GRID_Y);

        int idx = j * N_GRID_X + i;
        int idx_ip = j * N_GRID_X + ip;
        int idx_im = j * N_GRID_X + im;
        int idx_jp = jp * N_GRID_X + i;
        int idx_jm = jm * N_GRID_X + i;

        Ex[idx] = -(phi[idx_ip] - phi[idx_im]) / (2.0 * dx);
        Ey[idx] = -(phi[idx_jp] - phi[idx_jm]) / (2.0 * dy);
    }
}

__global__ void compute_rhs_kernel(const float_type* d_N, float_type* d_rhs, int NX, int NY, float_type dx, float_type dy) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= NX || j >= NY) return;

    int idx = j * NX + i;
    d_rhs[idx] = d_N[idx] / (dx * dy);

    // for debugging:
    //float_type x = i*dx;
    //float_type y = j*dy;
    //d_rhs[idx] = sin(2*M_PI*x) * sin(2*M_PI*y);

}

void solve_poisson_periodic(FieldContainer& fc) {
    // Compute residual
    // Block and grid config — safe for any N_GRID_X, N_GRID_Y
    int threadsPerBlockX = min(32, (int) (sqrt(threadsPerBlock))); // 32*32=1024 is max thread per block of T40 GPUs
    int threadsPerBlockY = min(32, (int) (sqrt(threadsPerBlock)));
    dim3 blockDim(threadsPerBlockX, threadsPerBlockY);
    dim3 gridDim(
        (N_GRID_X + blockDim.x - 1) / blockDim.x,
        (N_GRID_Y + blockDim.y - 1) / blockDim.y
    );

    int size = N_GRID_X * N_GRID_Y;
    float_type *d_rhs, *d_phi, *d_residual;

    CUDA_CHECK(hipMalloc(&d_rhs, size*sizeof(float_type)));
    CUDA_CHECK(hipMalloc(&d_phi, size*sizeof(float_type)));
    CUDA_CHECK(hipMalloc(&d_residual, size*sizeof(float_type)));

    //////////////////////////////
    // 1. Compute MC estimate
    //////////////////////////////

    // rhs
    compute_rhs_kernel<<<gridDim, blockDim>>>(fc.d_N, d_rhs, N_GRID_X, N_GRID_Y, dx, dy);
    hipDeviceSynchronize();

    float_type res_normalizer = compute_l2_norm(d_rhs, size); CUDA_CHECK(hipDeviceSynchronize());

    // solve
    poisson_fft_solver(N_GRID_X, N_GRID_Y, fc.dx, fc.dy, d_rhs, d_phi);

    // residual
    compute_residual_kernel<<<gridDim, blockDim>>>(d_phi, d_rhs, d_residual, N_GRID_X, N_GRID_Y, dx, dy);
    CUDA_CHECK(hipDeviceSynchronize());

    // compute L2 norm of residual
    float_type res_norm = compute_l2_norm(d_residual, size); CUDA_CHECK(hipDeviceSynchronize());
    res_norm /= res_normalizer;
    //printf("Residual L2 norm = %e\n", res_norm);

    // compute electric fied
    compute_electric_field_kernel_periodic<<<gridDim, blockDim>>>(d_phi, fc.d_Ex, fc.d_Ey, N_GRID_X, N_GRID_Y, dx, dy);
    hipDeviceSynchronize();

    //////////////////////////////
    // 2. Compute VR estimate
    //////////////////////////////

    // rhs
    compute_rhs_kernel<<<gridDim, blockDim>>>(fc.d_NVR, d_rhs, N_GRID_X, N_GRID_Y, dx, dy);
    hipDeviceSynchronize();

    res_normalizer = compute_l2_norm(d_rhs, size); CUDA_CHECK(hipDeviceSynchronize());

    // solve
    poisson_fft_solver(N_GRID_X, N_GRID_Y, fc.dx, fc.dy, d_rhs, d_phi);

    // residual
    compute_residual_kernel<<<gridDim, blockDim>>>(d_phi, d_rhs, d_residual, N_GRID_X, N_GRID_Y, dx, dy);
    CUDA_CHECK(hipDeviceSynchronize());

    // Compute L2 norm of residual
    res_norm = compute_l2_norm(d_residual, size); CUDA_CHECK(hipDeviceSynchronize());
    res_norm /= res_normalizer;
    //printf("Residual L2 norm = %e\n", res_norm);

    // compute electric fied
    compute_electric_field_kernel_periodic<<<gridDim, blockDim>>>(d_phi, fc.d_ExVR, fc.d_EyVR, N_GRID_X, N_GRID_Y, dx, dy);
    hipDeviceSynchronize();

    // Cleanup
    hipFree(d_phi);
    hipFree(d_residual);
    hipFree(d_rhs);
}
