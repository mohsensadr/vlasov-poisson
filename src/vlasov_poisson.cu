#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>

#include "constants.hpp"
#include "solver.cuh"
#include "initialization.cuh"
#include "IO.h"
#include "moments.cuh"

static __device__ int periodic_index(int i, int N) {
    return (i + N) % N;
}

__global__ void update_velocity_2d(float *x, float *y, float *vx, float *vy,
                                  float *Ex, float *Ey, int n_particles,
            int N_GRID_X, int N_GRID_Y,
            float Lx, float Ly,
            float DT,
            float Q_OVER_M) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_particles) return;

    float xi = x[i] / Lx * N_GRID_X;
    float yi = y[i] / Ly * N_GRID_Y;

    int ix = floorf(xi);
    int iy = floorf(yi);
    float dx = xi - ix;
    float dy = yi - iy;

    int ix0 = periodic_index(ix, N_GRID_X);
    int ix1 = periodic_index(ix + 1, N_GRID_X);
    int iy0 = periodic_index(iy, N_GRID_Y);
    int iy1 = periodic_index(iy + 1, N_GRID_Y);

    float w00 = (1 - dx) * (1 - dy);
    float w01 = (1 - dx) * dy;
    float w10 = dx * (1 - dy);
    float w11 = dx * dy;

    int i00 = ix0 + iy0 * N_GRID_X;
    int i01 = ix0 + iy1 * N_GRID_X;
    int i10 = ix1 + iy0 * N_GRID_X;
    int i11 = ix1 + iy1 * N_GRID_X;

    float Exi = w00 * Ex[i00] + w01 * Ex[i01] + w10 * Ex[i10] + w11 * Ex[i11];
    float Eyi = w00 * Ey[i00] + w01 * Ey[i01] + w10 * Ey[i10] + w11 * Ey[i11];

    vx[i] += - Q_OVER_M * Exi * DT;
    vy[i] += - Q_OVER_M * Eyi * DT;
}

__global__ void update_position_2d(float *x, float *y, float *vx, float *vy,
                                  int n_particles, float Lx, float Ly, float DT) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_particles) return;

    x[i] += vx[i] * DT;
    y[i] += vy[i] * DT;

    // periodic boundaries
    if (x[i] < 0) x[i] += Lx;
    if (x[i] >= Lx) x[i] -= Lx;
    if (y[i] < 0) y[i] += Ly;
    if (y[i] >= Ly) y[i] -= Ly;
}

__global__ void map_weights_2d(float *x, float *y, float *vx, float *vy, float *w,
    float *NVR, float *UxVR, float *UyVR, float *TVR, int n_particles,
    int N_GRID_X, int N_GRID_Y, float Lx, float Ly, bool global_to_local
    ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_particles) {
        float Navg = (1.0f*n_particles) / (1.0f*N_GRID_X*N_GRID_Y);
        int ix = int(x[i] / Lx * N_GRID_X) % N_GRID_X;
        int iy = int(y[i] / Ly * N_GRID_Y) % N_GRID_Y;
        int idx = ix + iy * N_GRID_X;
        float energy = (vx[i]-UxVR[idx])*(vx[i]-UxVR[idx]);
        energy += (vy[i]-UyVR[idx])*(vy[i]-UyVR[idx]);
        float energy0 = vx[i]*vx[i] + vy[i]*vy[i];
        float kbT_m = kb*TVR[idx]/m;
        float kbT_m0 = kb*1.0f/m;
        if(global_to_local){
          w[i] = w[i] * (NVR[idx]/Navg) * (kbT_m0/kbT_m) * expf(-energy/kbT_m/2.0f+energy0/kbT_m0/2.0f);
        }
        else{
          w[i] = w[i] * (Navg/NVR[idx]) * (kbT_m/kbT_m0) * expf(energy/kbT_m/2.0f-energy0/kbT_m0/2.0f);
        }
    }
}

void run(int N_GRID_X, int N_GRID_Y,
            int N_PARTICLES,
            float DT,
            int NSteps,
            float Lx,
            float Ly,
            int threadsPerBlock
            ) {
    hipMemcpyToSymbol(HIP_SYMBOL(kb), &kb_host, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(m), &m_host, sizeof(float));
    
    float dx = Lx/N_GRID_X;
    float dy = Ly/N_GRID_Y;
    int grid_size = N_GRID_X*N_GRID_Y;
    float *d_x, *d_y, *d_vx, *d_vy;
    float *d_N, *d_Ux, *d_Uy, *d_T, *d_Ex, *d_Ey;
    float *d_w, *d_NVR, *d_UxVR, *d_UyVR, *d_TVR;

    hipMalloc(&d_x, sizeof(float) * N_PARTICLES);
    hipMalloc(&d_y, sizeof(float) * N_PARTICLES);
    hipMalloc(&d_vx, sizeof(float) * N_PARTICLES);
    hipMalloc(&d_vy, sizeof(float) * N_PARTICLES);
    hipMalloc(&d_w, sizeof(float) * N_PARTICLES);

    hipMalloc(&d_N, sizeof(float) * N_GRID_X * N_GRID_Y);
    hipMalloc(&d_Ux, sizeof(float) * N_GRID_X * N_GRID_Y);
    hipMalloc(&d_Uy, sizeof(float) * N_GRID_X * N_GRID_Y);
    hipMalloc(&d_T, sizeof(float) * N_GRID_X * N_GRID_Y);
    hipMalloc(&d_Ex, sizeof(float) * N_GRID_X * N_GRID_Y);
    hipMalloc(&d_Ey, sizeof(float) * N_GRID_X * N_GRID_Y);

    hipMalloc(&d_NVR, sizeof(float) * N_GRID_X * N_GRID_Y);
    hipMalloc(&d_UxVR, sizeof(float) * N_GRID_X * N_GRID_Y);
    hipMalloc(&d_UyVR, sizeof(float) * N_GRID_X * N_GRID_Y);
    hipMalloc(&d_TVR, sizeof(float) * N_GRID_X * N_GRID_Y);

    int blocksPerGrid = (N_PARTICLES + threadsPerBlock - 1) / threadsPerBlock;

    // initialize particle velocity and position
    initialize_particles<<<blocksPerGrid, threadsPerBlock>>>(
        d_x, d_y, d_vx, d_vy, Lx, Ly, N_PARTICLES
    );
    hipDeviceSynchronize();

    // compute moments, needed to find emperical density field
    compute_moments(d_x, d_y, d_vx, d_vy, d_N, d_Ux, d_Uy, d_T, d_w, d_NVR, d_UxVR, d_UyVR, d_TVR,
        N_PARTICLES, N_GRID_X, N_GRID_Y, Lx, Ly, blocksPerGrid, threadsPerBlock);
    hipDeviceSynchronize();

    // set particle weights given estimted and exact fields
    initialize_weights<<<blocksPerGrid, threadsPerBlock>>>(
        d_x, d_y, d_N, d_w, N_PARTICLES, N_GRID_X, N_GRID_Y, Lx, Ly
    );
    hipDeviceSynchronize();

    // recompute moments given weights, mainly for VR estimate
    compute_moments(d_x, d_y, d_vx, d_vy, d_N, d_Ux, d_Uy, d_T, d_w, d_NVR, d_UxVR, d_UyVR, d_TVR,
        N_PARTICLES, N_GRID_X, N_GRID_Y, Lx, Ly, blocksPerGrid, threadsPerBlock);
    hipDeviceSynchronize();

    // write out initial fields
    post_proc(d_N, d_Ux, d_Uy, d_T, d_NVR, d_UxVR, d_UyVR, d_TVR, grid_size, 0);
    hipDeviceSynchronize();

    for (int step = 1; step < NSteps+1; ++step) {

        // compute Electric field
        solve_poisson_jacobi(d_N, d_Ex, d_Ey, N_GRID_X, N_GRID_Y, dx, dy, threadsPerBlock);
        hipDeviceSynchronize();

        // map weights from global to local eq.
        map_weights_2d<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, d_vx, d_vy, d_w, d_NVR, d_UxVR, d_UyVR, d_TVR, N_PARTICLES, N_GRID_X, N_GRID_Y,
            Lx, Ly, true);
        hipDeviceSynchronize();

        // push particles in the velocity space
        update_velocity_2d<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, d_vx, d_vy, d_Ex, d_Ey, N_PARTICLES, N_GRID_X, N_GRID_Y,
            Lx, Ly, DT, Q_OVER_M);
        hipDeviceSynchronize();

        // map weights from local to global eq.
        map_weights_2d<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, d_vx, d_vy, d_w, d_NVR, d_UxVR, d_UyVR, d_TVR, N_PARTICLES, N_GRID_X, N_GRID_Y,
            Lx, Ly, false);
        hipDeviceSynchronize();

        // push particles in the position space
        update_position_2d<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, d_vx, d_vy, N_PARTICLES, Lx, Ly, DT);
        hipDeviceSynchronize();

        // update moments
        compute_moments(d_x, d_y, d_vx, d_vy, d_N, d_Ux, d_Uy, d_T, d_w, d_NVR, d_UxVR, d_UyVR, d_TVR,
            N_PARTICLES, N_GRID_X, N_GRID_Y, Lx, Ly, blocksPerGrid, threadsPerBlock);
        hipDeviceSynchronize();

        // print output
        if (step % 10 == 0) {
            post_proc(d_N, d_Ux, d_Uy, d_T, d_NVR, d_UxVR, d_UyVR, d_TVR, grid_size, step);
            hipDeviceSynchronize();
        }
    }

    // free memory
    hipFree(d_x); hipFree(d_y); hipFree(d_vx); hipFree(d_vy);
    hipFree(d_N); hipFree(d_Ux); hipFree(d_Uy);hipFree(d_T);
    hipFree(d_Ex); hipFree(d_Ey);
    hipFree(d_w); hipFree(d_NVR); hipFree(d_UxVR); hipFree(d_UyVR);
    hipFree(d_TVR);

    std::cout << "Done.\n";
}

