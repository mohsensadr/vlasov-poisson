#include "constants.hpp"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include "moments.cuh"
#include "Depositors/BruteDepositor.h"
#include "Depositors/TiledDepositor.h"
#include "Depositors/SortedDepositor.h"

void compute_moments(ParticleContainer& pc, FieldContainer& fc, Sorting& sorter) {
    std::unique_ptr<MomentDepositor> depositor;

    switch (depositionMode) {
        case DepositionMode::BRUTE:   depositor = std::make_unique<BruteDepositor>(); break;
        case DepositionMode::TILING:  depositor = std::make_unique<TiledDepositor>(); break;
        case DepositionMode::SORTING: depositor = std::make_unique<SortingDepositor>(sorter); break;
    }

    fc.setZero();
    hipMemcpyToSymbol(HIP_SYMBOL(kb), &kb_host, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(m), &m_host, sizeof(float));

    depositor->deposit(pc, fc, sorter);
}