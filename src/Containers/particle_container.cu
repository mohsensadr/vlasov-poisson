#include "hip/hip_runtime.h"
// particle_container.cu
#include "particle_container.cuh"
#include <cmath>

ParticleContainer::ParticleContainer(int n_particles_) : n_particles(n_particles_) {
    size_t bytes = n_particles * sizeof(float);
    hipMalloc(&d_x, bytes);
    hipMalloc(&d_y, bytes);
    hipMalloc(&d_vx, bytes);
    hipMalloc(&d_vy, bytes);
    hipMalloc(&d_w, bytes);
    hipMalloc(&d_wold, bytes);
}

ParticleContainer::~ParticleContainer() {
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_vx);
    hipFree(d_vy);
    hipFree(d_w);
    hipFree(d_wold);
}

__device__ int periodic_index(int i, int N) {
    return (i + N) % N;
}

void ParticleContainer::update_velocity(float *Ex, float *Ey,
                                        int N_GRID_X, int N_GRID_Y,
                                        float Lx, float Ly,
                                        float DT, float Q_OVER_M) 
{
    update_velocity_2d<<<blocksPerGrid, threadsPerBlock>>>(
        d_x, d_y, d_vx, d_vy, Ex, Ey, n_particles,
        N_GRID_X, N_GRID_Y, Lx, Ly, DT, Q_OVER_M
    );
    hipDeviceSynchronize();
}

void ParticleContainer::update_position(float Lx, float Ly, float DT) {
    update_position_2d<<<blocksPerGrid, threadsPerBlock>>>(
        d_x, d_y, d_vx, d_vy, n_particles, Lx, Ly, DT
    );
    hipDeviceSynchronize();
}

void ParticleContainer::map_weights(float *NVR, float *UxVR, float *UyVR, float *TVR,
                                    int N_GRID_X, int N_GRID_Y, float Lx, float Ly, bool global_to_local)
{
    map_weights_2d<<<blocksPerGrid, threadsPerBlock>>>(
        d_x, d_y, d_vx, d_vy, d_w, NVR, UxVR, UyVR, TVR, n_particles,
        N_GRID_X, N_GRID_Y, Lx, Ly, global_to_local
    );
    hipDeviceSynchronize();
}

__global__ void update_velocity_2d(float *x, float *y, float *vx, float *vy,
                                  float *Ex, float *Ey, int n_particles,
            int N_GRID_X, int N_GRID_Y,
            float Lx, float Ly,
            float DT,
            float Q_OVER_M) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_particles) return;

    float xi = x[i] / Lx * N_GRID_X;
    float yi = y[i] / Ly * N_GRID_Y;

    int ix = floorf(xi);
    int iy = floorf(yi);
    float dx = xi - ix;
    float dy = yi - iy;

    int ix0 = periodic_index(ix, N_GRID_X);
    int ix1 = periodic_index(ix + 1, N_GRID_X);
    int iy0 = periodic_index(iy, N_GRID_Y);
    int iy1 = periodic_index(iy + 1, N_GRID_Y);

    float w00 = (1 - dx) * (1 - dy);
    float w01 = (1 - dx) * dy;
    float w10 = dx * (1 - dy);
    float w11 = dx * dy;

    int i00 = ix0 + iy0 * N_GRID_X;
    int i01 = ix0 + iy1 * N_GRID_X;
    int i10 = ix1 + iy0 * N_GRID_X;
    int i11 = ix1 + iy1 * N_GRID_X;

    float Exi = w00 * Ex[i00] + w01 * Ex[i01] + w10 * Ex[i10] + w11 * Ex[i11];
    float Eyi = w00 * Ey[i00] + w01 * Ey[i01] + w10 * Ey[i10] + w11 * Ey[i11];

    vx[i] += - Q_OVER_M * Exi * DT;
    vy[i] += - Q_OVER_M * Eyi * DT;
}

__global__ void update_position_2d(float *x, float *y, float *vx, float *vy,
                                  int n_particles, float Lx, float Ly, float DT) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_particles) return;

    x[i] += vx[i] * DT;
    y[i] += vy[i] * DT;

    // periodic boundaries
    if (x[i] < 0) x[i] += Lx;
    if (x[i] >= Lx) x[i] -= Lx;
    if (y[i] < 0) y[i] += Ly;
    if (y[i] >= Ly) y[i] -= Ly;
}

__global__ void map_weights_2d(float *x, float *y, float *vx, float *vy, float *w,
    float *NVR, float *UxVR, float *UyVR, float *TVR, int n_particles,
    int N_GRID_X, int N_GRID_Y, float Lx, float Ly, bool global_to_local
    ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_particles) {
        float Navg = (1.0f*n_particles) / (1.0f*N_GRID_X*N_GRID_Y);
        int ix = int(x[i] / Lx * N_GRID_X) % N_GRID_X;
        int iy = int(y[i] / Ly * N_GRID_Y) % N_GRID_Y;
        int idx = ix + iy * N_GRID_X;
        float energy = (vx[i]-UxVR[idx])*(vx[i]-UxVR[idx]);
        energy += (vy[i]-UyVR[idx])*(vy[i]-UyVR[idx]);
        float energy0 = vx[i]*vx[i] + vy[i]*vy[i];
        float kbT_m = TVR[idx]; //kb/m=1
        float kbT_m0 = 1.0f; // kb/m=1;
        if(global_to_local){
          w[i] = w[i] * (NVR[idx]/Navg) * (kbT_m0/kbT_m) * expf(-energy/kbT_m/2.0f+energy0/kbT_m0/2.0f);
        }
        else{
          w[i] = w[i] * (Navg/NVR[idx]) * (kbT_m/kbT_m0) * expf(energy/kbT_m/2.0f-energy0/kbT_m0/2.0f);
        }
    }
}