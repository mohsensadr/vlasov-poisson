#include <iostream>
#include <fstream>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>

#include "Constants/constants.hpp"
#include "Solvers/solver.cuh"
#include "Initializations/initialization.cuh"
#include "IOs/IO.h"
#include "Depositors/moments.cuh"
#include "Containers/particle_container.cuh"
#include "Containers/field_container.cuh"
#include "Distributions/pdfs.cuh"
#include "Sorters/sorting.cuh"
#include "VRs/MxE.cuh"

void run(const std::string& pdf_type, float* pdf_params) {
    hipMemcpyToSymbol(HIP_SYMBOL(kb), &kb_host, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(m), &m_host, sizeof(float));

    // TODO: dx, dy, Lx, Ly are member variables of field container, remove them from here.
    dx = Lx/N_GRID_X;
    dy = Ly/N_GRID_Y;
    grid_size = N_GRID_X*N_GRID_Y;

    ParticleContainer pc(N_PARTICLES);
    FieldContainer fc(N_GRID_X, N_GRID_Y, Lx, Ly);
    Sorting sorter(pc, fc);

    // Create the appropriate PDF struct for device use
    PDF_position pdf_position;
    if (pdf_type == "gaussian" || pdf_type == "Gaussian") {
        pdf_position = make_gaussian_pdf(pdf_params[0], Lx, Ly);
    } else if (pdf_type == "cosine" || pdf_type == "Cosine") {
        pdf_position = make_cosine_pdf(pdf_params[0], pdf_params[1], Lx, Ly);
    } else if (pdf_type == "double_gaussian" || pdf_type == "DoubleGaussian") {
        pdf_position = make_double_gaussian_pdf(pdf_params[0], pdf_params[1], pdf_params[2], pdf_params[3], 
                                              pdf_params[4], pdf_params[5], pdf_params[6], pdf_params[7], Lx, Ly);
    } else {
        throw std::invalid_argument("Unknown PDF type: " + pdf_type);
    }

    // initialize particle velocity and position
    initialize_particles<<<blocksPerGrid, threadsPerBlock>>>(
        pc.d_x, pc.d_y, pc.d_vx, pc.d_vy, Lx, Ly, N_PARTICLES, pdf_position
    );
    hipDeviceSynchronize();

    // compute moments, needed to find emperical density field
    if (depositionMode == DepositionMode::SORTING) {
      sorter.sort_particles_by_cell();
    }
    compute_moments(pc, fc, sorter);
    hipDeviceSynchronize();

    // set particle weights given estimted and exact fields
    initialize_weights<<<blocksPerGrid, threadsPerBlock>>>(
        pc.d_x, pc.d_y, fc.d_N, pc.d_w, N_PARTICLES, N_GRID_X, N_GRID_Y, Lx, Ly, pdf_position
    );
    hipDeviceSynchronize();

    // recompute moments given weights, mainly for VR estimate
    if (depositionMode == DepositionMode::SORTING) {
      sorter.sort_particles_by_cell();
    }
    compute_moments(pc, fc, sorter);
    hipDeviceSynchronize();

    // write out initial fields
    post_proc(fc, 0);
    hipDeviceSynchronize();

    size_t size = N_PARTICLES * sizeof(float);

    for (int step = 1; step < NSteps+1; ++step) {
        // compute Electric field
        solve_poisson_periodic(fc);
        hipDeviceSynchronize();

        // update wold given w
        hipMemcpy(pc.d_wold, pc.d_w, size, hipMemcpyDeviceToDevice);
        hipDeviceSynchronize();

        // map weights from global to local eq.
        pc.map_weights(fc.d_NVR, fc.d_UxVR, fc.d_UyVR, fc.d_TVR, N_GRID_X, N_GRID_Y, Lx, Ly, true);
        hipDeviceSynchronize();

        // Push particles in the velocity space
        // Use either MC or VR density estimtes in the rhs of the Poisson to get E
        if (rhsMode == RhsMode::VR)
          pc.update_velocity(fc.d_ExVR, fc.d_EyVR, N_GRID_X, N_GRID_Y, Lx, Ly, DT, Q_OVER_M);
        else
          pc.update_velocity(fc.d_Ex, fc.d_Ey, N_GRID_X, N_GRID_Y, Lx, Ly, DT, Q_OVER_M);
        hipDeviceSynchronize();

        // map weights from local to global eq.
        pc.map_weights(fc.d_NVR, fc.d_UxVR, fc.d_UyVR, fc.d_TVR, N_GRID_X, N_GRID_Y, Lx, Ly, false);
        hipDeviceSynchronize();

        // MxE to conserve equil. moments.
        if (vrMode == VRMode::MXE) {
          update_weights_dispatch(pc.d_vx, pc.d_vy, sorter.d_cell_offsets, pc.d_w, pc.d_wold, fc.d_NVR, fc.d_UxVR, fc.d_UyVR, fc.d_ExVR, fc.d_EyVR, grid_size, Nm);
          hipDeviceSynchronize();
        }
        
        // push particles in the position space
        pc.update_position(Lx, Ly, DT);
        hipDeviceSynchronize();

        // update moments
        if (depositionMode == DepositionMode::SORTING) {
          sorter.sort_particles_by_cell();
          hipDeviceSynchronize();
        }

        compute_moments(pc, fc, sorter);
        hipDeviceSynchronize();

        // print output
        if (step % 10 == 0) {
            post_proc(fc, step);
            hipDeviceSynchronize();
        }
    }

    std::cout << "Done.\n";
}

