#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "VRs/MxE.cuh"
#include "Constants/constants.hpp"

template<int Nm>
__device__ void Gauss_Jordan(float_type H[Nm][Nm], float_type g[Nm], float_type x[Nm]) {
    for (int i = 0; i < Nm; i++) {
        float_type diag = H[i][i];
        for (int j = i; j < Nm; j++) H[i][j] /= diag;
        g[i] /= diag;

        for (int k = 0; k < Nm; k++) {
            if (k == i) continue;
            float_type factor = H[k][i];
            for (int j = i; j < Nm; j++) H[k][j] -= factor * H[i][j];
            g[k] -= factor * g[i];
        }
    }

    for (int i = 0; i < Nm; i++) x[i] = g[i];
}

template<int Nm>
__device__ float_type mom(float_type u1, float_type u2, float_type U_1, float_type U_2, int n) {
    switch(n) {
        case 0: return u1 - U_1;
        case 1: return u2 - U_2;
        case 2: return (u1 - U_1) * (u1 - U_1) + (u2 - U_2) * (u2 - U_2);
    }
    return 0.0;
}

template<int Nm>
__global__ void update_weights(
    const float_type* __restrict__ vx,
    const float_type* __restrict__ vy,
    const int* __restrict__ d_cell_offsets,
    float_type* __restrict__ w,
    float_type* __restrict__ wold,
    float_type* __restrict__ NVR,
    float_type* __restrict__ UxVR,
    float_type* __restrict__ UyVR,
    float_type* __restrict__ ExVR,
    float_type* __restrict__ EyVR,
    int num_cells,
    int n_particles,
    float_type dt
) {
    int cell = blockIdx.x * blockDim.x + threadIdx.x;
    if (cell >= num_cells) return;

    float_type Navg = float_type(n_particles) / float_type(num_cells);
    float_type tol = Tolerance<float_type>::value();
    int start = d_cell_offsets[cell];
    int end   = d_cell_offsets[cell + 1];
    int Npc = end - start;

    if (Npc < 100) return;

    float_type p[Nm] = {0.0};
    float_type pt[Nm] = {0.0};
    float_type p0[Nm] = {0.0};
    p0[2] = 2.0;

    float_type sumwold = 0.0;

    for (int i = start; i < end; i++) {
        sumwold += wold[i];
        for (int j = 0; j < Nm; j++) {
            p[j] += mom<Nm>(vx[i], vy[i], 0.0, 0.0, j);
            pt[j] += (1.0 - wold[i]) * mom<Nm>(vx[i], vy[i], 0.0, 0.0, j);
        }
    }

    for (int i = 0; i < Nm; i++) {
        p[i] /= Npc;
        pt[i] /= NVR[cell];
        //printf("p[%d]=%f | ", i, p[i]);
        //printf("inter1 pt[%d]=%f | ", i, pt[i]);
    }

    for (int i = 0; i < Nm; i++) {
        pt[i] += p0[i];
        //printf("inter2 pt[%d]=%f |", i, pt[i]);
    }

    // correct moments using Ex and Ey
    pt[0] -= dt * ExVR[cell];
    pt[1] -= dt * EyVR[cell];
    pt[2] -= dt * (UxVR[cell]*ExVR[cell] + UyVR[cell]*EyVR[cell]);

    //for (int i = 0; i < Nm; i++) {
    //    printf("inter3 pt[%d]=%f | ", i, pt[i]);
    //}
    //printf("\n\n dt * ExVR[%d]=%f | ", cell, dt * ExVR[cell]);
    //printf("dt * EyVR[%d]=%f | ", cell, dt * EyVR[cell]);
    //printf("\ndt * (UxVR[%d]*ExVR[.] + UyVR[.]*EyVR[.]) = %f", cell, dt * (UxVR[cell]*ExVR[cell] + UyVR[cell]*EyVR[cell]));

    // now compute target <w*R(v)> moments: 
    // <R(v)>VR = <R(v)>0 + <(1-w)*R(v)> 
    // = <R(v)>0 + <R(v)> - <w*R(v)> 
    // which implies: <w*R(v)> = <R(v)>0 + <R(v)> - <R(v)>VR 
    // here we reuse variable p to denote <w*R(v)> from this point on
    for (int i = 0; i < Nm; i++) {
        p[i] = p0[i] + p[i] - pt[i];
        //printf("\n\n final target moment p[%d]=%f |", i, p[i]);
    }

    for (int i = start; i < end; i++)
        wold[i] = w[i];

    bool convergence = false;
    int max_iter = 1000;
    int iter = 0;
    float_type g[Nm], H[Nm][Nm], xvec[Nm], lam[Nm] = {0.0};

    while (!convergence) {
        iter++;
        if (iter > max_iter) break;

        // Compute gradient
        float_type res = 0.0;
        for (int j = 0; j < Nm; j++) {
            g[j] = 0.0;
            for (int i = start; i < end; i++)
                g[j] += w[i] * mom<Nm>(vx[i], vy[i], UxVR[cell], UyVR[cell], j);
            g[j] = g[j]/Npc - p[j];
            res += fabsf(g[j]);
        }
        if (res < tol){
          convergence = true;
          break;
        }

        // Compute Hessian
        for (int i = 0; i < Nm; i++)
            for (int j = 0; j < Nm; j++)
                H[i][j] = 0.0;

        for (int k = 0; k < Nm; k++) {
            for (int j = k; j < Nm; j++) {
                float_type Ski = 0.0, Sji = 0.0, SkiSji = 0.0;
                for (int i = start; i < end; i++) {
                    float_type mk = mom<Nm>(vx[i], vy[i], UxVR[cell], UyVR[cell], k);
                    float_type mj = mom<Nm>(vx[i], vy[i], UxVR[cell], UyVR[cell], j);
                    Ski += mk * w[i];
                    Sji += mj * w[i];
                    SkiSji += mk * mj * w[i];
                }
                H[k][j] = SkiSji/Npc - Ski/Npc*p[j] - Sji/Npc*p[k] + p[j]*p[k];
            }
        }

        for (int k = 0; k < Nm; k++)
            for (int j = 0; j < k; j++)
                H[k][j] = H[j][k];

        // Solve for Newton step
        Gauss_Jordan<Nm>(H, g, xvec);

        // Update weights
        float_type sumW = 0.0;
        for (int j = 0; j < Nm; j++)
            lam[j] -= xvec[j];

        for (int i = start; i < end; i++) {
            float_type dummy = 0.0;
            for (int j = 0; j < Nm; j++)
                dummy += lam[j]*(mom<Nm>(vx[i], vy[i], UxVR[cell], UyVR[cell], j)-p[j]);
            float_type dummy2 = expf(-dummy);
            w[i] = wold[i] / dummy2;
            sumW += w[i];
        }

        for (int i = start; i < end; i++)
            w[i] *= sumwold/sumW;
    }
    if(!convergence){
      for (int i = start; i < end; i++){
        w[i] = wold[i];
      }
    }
    if(iter > 999)
      printf("MxE iter %d in cell %d\n", iter, cell);
}

void update_weights_dispatch(
    const float_type* vx,
    const float_type* vy,
    const int* d_cell_offsets,
    float_type* w,
    float_type* wold,
    float_type* NVR,
    float_type* UxVR,
    float_type* UyVR,
    float_type* ExVR,
    float_type* EyVR,
    int num_cells,
    int Nm
) {

    switch (Nm) {
        case 3:
            update_weights<3><<<blocksPerGrid, threadsPerBlock>>>(vx, vy, d_cell_offsets, w, wold, NVR, UxVR, UyVR, ExVR, EyVR, num_cells, N_PARTICLES, DT);
            break;
        // Add more cases as needed
        default:
            printf("Unsupported Nm: %d\n", Nm);
            break;
    }
}
